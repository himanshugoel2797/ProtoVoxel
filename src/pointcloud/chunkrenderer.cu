#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#define GLM_FORCE_CUDA
#include "glm/glm.hpp"
#include "chunkmanager.h"

namespace PPC = ProtoVoxel::PointCloud;
namespace cg = cooperative_groups;

typedef struct
{
        glm::mat4 proj;
        glm::mat4 view;
        glm::mat4 vp;
        glm::mat4 ivp;
        glm::mat4 prev_proj;
        glm::mat4 prev_view;
        glm::mat4 prev_vp;
        glm::mat4 prev_ivp;
        glm::vec4 prev_eyePos;
        glm::vec4 prev_eyeUp;
        glm::vec4 prev_eyeDir;
        glm::vec4 eyePos;
        glm::vec4 eyeUp;
        glm::vec4 eyeDir;
        glm::vec4 eyeRight;
} globalParams_t;

__device__ __forceinline__ int floatToOrderedInt(float floatVal) {
    int intVal = __float_as_int(floatVal);
    return (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF;
}
__device__ __forceinline__ float orderedIntToFloat(int intVal) {
    return __int_as_float((intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF);
}

__global__ void splat_kernel(void* global_vars, glm::uvec4* splat_points, uint32_t* pointBuffer, hipSurfaceObject_t colorBuffer, int w, int h, int draw_count)
{

    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    globalParams_t* globalParams = (globalParams_t*)global_vars;

    if(ptIdx >= draw_count)
        return;

    glm::uvec4 data = splat_points[ptIdx];

    glm::vec4 ppos = glm::vec4(data.x / (float)POS_SCALE_FACTOR, data.y / (float)POS_SCALE_FACTOR, data.z / (float)POS_SCALE_FACTOR, 1.0f);
    uchar4 pcol = make_uchar4((data.w & 0xff), ((data.w >> 8) & 0xff), ((data.w >> 16) & 0xff), ((data.w >> 24) & 0xff));

    ppos = globalParams->vp * ppos;
    ppos /= ppos.w;
    
    ppos.x = (ppos.x + 1.0f) * 0.5f;
    ppos.y = (ppos.y + 1.0f) * 0.5f;
    glm::ivec2 ppos_pxl = glm::ivec2(__float2int_rd(ppos.x * w), __float2int_rd(ppos.y * h));

    if (ppos_pxl.x < 0)
        return;
    if (ppos_pxl.x >= w)
        return;
    if (ppos_pxl.y < 0.0f)
        return;
    if (ppos_pxl.y >= h)
        return;
    if (ppos.z < 0)
        return;

    int cur_depth = floatToOrderedInt(ppos.z);

    uint32_t pointBufferIdx = ppos_pxl.y * w + ppos_pxl.x;
    //get all subgroups where the point is in the same pixel

    unsigned int mask = __match_any_sync(__activemask(), pointBufferIdx);
    /*if (__popc(mask) <= 1)
    {
        if(cur_depth > atomicMax(&pointBuffer[pointBufferIdx], cur_depth))
        {
            surf2Dwrite(pcol, colorBuffer, ppos_pxl.x * sizeof(uchar4), ppos_pxl.y);
        }
    }else*/
    {
        if(cur_depth == __reduce_max_sync(mask, cur_depth))
        {
            if(cur_depth > atomicMax(&pointBuffer[pointBufferIdx], cur_depth))
            {
                //pcol = make_uchar4((int)(ppos.z / 0.01f * 255), (int)(ppos.z / 0.01f * 255), (int)(ppos.z / 0.01f * 255), (int)(ppos.z / 0.01f * 255));
                //printf("%d %f\n", mask, ppos.z);
                surf2Dwrite(pcol, colorBuffer, ppos_pxl.x * sizeof(uchar4), ppos_pxl.y);
            }
        }
    }

}

void PPC::ChunkManager::splat(void* global_vars, void* splat_points, void* pointBuffer, hipSurfaceObject_t colorBuffer, int w, int h, int draw_count)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (draw_count + threadsPerBlock - 1) / threadsPerBlock;
    splat_kernel<<<blocksPerGrid, threadsPerBlock>>>(global_vars, (glm::uvec4*)splat_points, (uint32_t*)pointBuffer, colorBuffer, w, h, draw_count);
}